#include "hip/hip_runtime.h"
// C++ program to create target string, starting from 
// random string using Genetic Algorithm 

#include <bits/stdc++.h> 
#include <chrono>

using namespace std; 

// Number of individuals in each generation 
#define POPULATION_SIZE 100 

// Valid Genes 
const string GENES = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ 1234567890, .-;:_!\"#%&/()=?@${[]}"; 

// Target string to be generated 
const string TARGET = "I love GeeksforGeeksI love GeeksforGeeksI love GeeksforGeeks"; 

// Function to generate random numbers in given range 
int random_num(int start, int end) 
{ 
	int range = (end-start)+1; 
	int random_int = start+(rand()%range); 
	return random_int; 
} 

// Create random genes for mutation 
char mutated_genes() 
{ 
	int len = GENES.size(); 
	int r = random_num(0, len-1); 
	return GENES[r]; 
} 

// create chromosome or string of genes 
string create_gnome() 
{ 
	int len = TARGET.size(); 
	string gnome = ""; 
	for(int i = 0;i<len;i++) 
		gnome += mutated_genes(); 
	return gnome; 
} 

// Class representing individual in population 
class Individual 
{ 
public: 
	string chromosome; 
	int fitness; 
	Individual(string chromosome); 
	Individual mate(Individual parent2); 
	int cal_fitness(); 
}; 

Individual::Individual(string chromosome) 
{ 
	this->chromosome = chromosome; 
	fitness = cal_fitness(); 
}; 

// Perform mating and produce new offspring 
Individual Individual::mate(Individual par2) 
{ 
	// chromosome for offspring 
	string child_chromosome = ""; 

	int len = chromosome.size(); 
	for(int i = 0;i<len;i++) 
	{ 
		// random probability 
		float p = random_num(0, 100)/100; 

		// if prob is less than 0.45, insert gene 
		// from parent 1 
		if(p < 0.45) 
			child_chromosome += chromosome[i]; 

		// if prob is between 0.45 and 0.90, insert 
		// gene from parent 2 
		else if(p < 0.90) 
			child_chromosome += par2.chromosome[i]; 

		// otherwise insert random gene(mutate), 
		// for maintaining diversity 
		else
			child_chromosome += mutated_genes(); 
	} 

	// create new Individual(offspring) using 
	// generated chromosome for offspring 
	return Individual(child_chromosome); 
}; 


// Calculate fittness score, it is the number of 
// characters in string which differ from target 
// string. 
int Individual::cal_fitness() 
{ 
	int len = TARGET.size(); 
	int fitness = 0; 
	for(int i = 0;i<len;i++) 
	{ 
		if(chromosome[i] != TARGET[i]) 
			fitness++; 
	} 
	return fitness;	 
}; 

// Overloading < operator 
bool operator<(const Individual &ind1, const Individual &ind2) 
{ 
	return ind1.fitness < ind2.fitness; 
} 

__global__ void gpu_mate(const char* parent1, const char* parent2, char* offspring, float* probability, char* mutated) {
    int i = threadIdx.x;
	//printf("index:%d, %f, %c\n", i, probability[i], mutated[i]);

	if(probability[i] < 0.45)
		offspring[i] = parent1[i];
	else if(probability[i] < 0.9)
		offspring[i] = parent2[i];
	else
		offspring[i] = mutated[i];
}


string convertToString(char* a, int size) 
{ 
    int i; 
    string s = ""; 
    for (i = 0; i < size; i++) { 
        s = s + a[i]; 
    } 
    return s; 
} 

// Driver code 
int main(int argc, char **argv) 
{ 	
	int gpu = 0;
	if(argv[1] != NULL && (strcmp(argv[1],"gpu") == 0))
		gpu = 1;

	srand((unsigned)(time(0))); 

	// current generation 
	int generation = 0; 

	vector<Individual> population; 
	bool found = false; 

	// create initial population 
	for(int i = 0;i<POPULATION_SIZE;i++) 
	{ 
		string gnome = create_gnome(); 
		population.push_back(Individual(gnome)); 
	} 

	auto start = std::chrono::system_clock::now();

	char* gpu_parent1, *gpu_parent2, *gpu_offspring, *gpu_mutated;
	float* gpu_probability;

	if(gpu)
	{
		// Allocate device memory 
		hipMalloc((void**)&gpu_parent1, sizeof(char) * TARGET.size());
		hipMalloc((void**)&gpu_parent2, sizeof(char) * TARGET.size());
		hipMalloc((void**)&gpu_offspring, sizeof(char) * TARGET.size());
		hipMalloc((void**)&gpu_probability, sizeof(float) * TARGET.size());
		hipMalloc((void**)&gpu_mutated, sizeof(char) * TARGET.size());
	}

	while(! found) 
	{ 

		// sort the population in increasing order of fitness score 
		sort(population.begin(), population.end()); 

		// if the individual having lowest fitness score ie. 
		// 0 then we know that we have reached to the target 
		// and break the loop 
		if(population[0].fitness <= 0) 
		{ 
			found = true; 
			break; 
		} 

		// Otherwise generate new offsprings for new generation 
		vector<Individual> new_generation; 

		// Perform Elitism, that mean 10% of fittest population 
		// goes to the next generation 
		int s = (10*POPULATION_SIZE)/100; 
		for(int i = 0;i<s;i++) 
			new_generation.push_back(population[i]); 

		// From 50% of fittest population, Individuals 
		// will mate to produce offspring 
		s = (90*POPULATION_SIZE)/100; 

		for(int i = 0;i<s;i++) 
		{ 
			int len = population.size(); 
			int r = random_num(0, 50); 
			Individual parent1 = population[r]; 
			r = random_num(0, 50); 
			Individual parent2 = population[r]; 

			// using gpu
			if(gpu)
			{
				char offspring[TARGET.size()]={0};
				float probability[TARGET.size()]={0.0};
				char mutated[TARGET.size()] = {0};
	
				for(int i=0;i<TARGET.size();i++)
				{
					probability[i] = (float)random_num(0,100)/100;
					mutated[i] = GENES[random_num(0,GENES.size())];
					//cout<<"probability[i]:"<<probability[i]<<endl;
					//cout<<"mutated[i]:"<<mutated[i]<<endl;
				}

				// Transfer data from host to device memory
				hipMemcpy(gpu_parent1, parent1.chromosome.c_str(), sizeof(char) * TARGET.size(), hipMemcpyHostToDevice);
				hipMemcpy(gpu_parent2, parent2.chromosome.c_str(), sizeof(char) * TARGET.size(), hipMemcpyHostToDevice);
				hipMemcpy(gpu_mutated, mutated, sizeof(char) * TARGET.size(), hipMemcpyHostToDevice);
				hipMemcpy(gpu_probability, probability, sizeof(float) * TARGET.size(), hipMemcpyHostToDevice);

				gpu_mate<<<1,TARGET.size()>>>(gpu_parent1, gpu_parent2, gpu_offspring, gpu_probability, gpu_mutated);

				// Transfer data back to host memory
				hipMemcpy(offspring, gpu_offspring, sizeof(char) * TARGET.size(), hipMemcpyDeviceToHost);
				new_generation.push_back(convertToString(offspring, TARGET.size())); 
			}
			else
			{
				Individual offspring = parent1.mate(parent2); 
				new_generation.push_back(offspring); 
			}
			// end
		} 
		population = new_generation; 
		cout<< "Generation: " << generation << "\t"; 
		cout<< "String: "<< population[0].chromosome <<"\t"; 
		cout<< "Fitness: "<< population[0].fitness << "\n"; 

		generation++; 
	} 
    auto end = chrono::system_clock::now();
    chrono::duration<double> elapsed_seconds = end-start;
    time_t end_time = chrono::system_clock::to_time_t(end);
    cout << "finished computation at " << ctime(&end_time)
              << "elapsed time: " << elapsed_seconds.count() << "s\n";

    float cost_per_generation = elapsed_seconds.count()/generation;
    cout<<"cost_per_generation:"<<cost_per_generation<<endl;

	cout<< "Generation: " << generation << "\t"; 
	cout<< "String: "<< population[0].chromosome <<"\t"; 
	cout<< "Fitness: "<< population[0].fitness << "\n"; 

	if(gpu)
	{
		// Deallocate device memory
		hipFree(gpu_parent1);
		hipFree(gpu_parent2);
		hipFree(gpu_probability);
		hipFree(gpu_offspring);
		hipFree(gpu_mutated);
	}
} 
